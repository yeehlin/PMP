#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

__device__ hipExtent devextent;
__device__ hipPitchedPtr devPitchedPtr;

// ����������ԭʼ��������
typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture// 3D����
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture//ת���������

//__device__ hipPitchedPtr d_PitchedPtr;

typedef struct
{
    float4 m[3];
} float3x4;

// �ӽǾ���λ��GPU�ĳ����洢��
__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

// ������ߣ���ʼλ�úͷ���
struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

// intersect ray with a box���� �ͺ����ཻ
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(
Ray r, // ����
float3 boxmin,//  
float3 boxmax, //
float *tnear, // ������
float *tfar // Զ����
) 
{
	// ������������������ཻ�Ľ���
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// ͨ�������ӽǾ��󣩱任������������ת����
// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
	// dot��ʾ��ˣ�Ҳ�����������������ڻ�
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]����һ���������ݹ���0��1֮��
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);

    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__global__ void
d_render(
uint *d_output, //�������
uint imageW, //ͼ��Ŀ��
uint imageH,//ͼ��ĸ߶�
         float density, //ͼ���ܶ�
		 float brightness,//ͼ������
         float transferOffset, //����
		 float transferScale//�����Ϳ̶�
		 )
{
    const int maxSteps = 500;
    const float tstep = 0.01f;// ����
    const float opacityThreshold = 0.95f;// ͸���ȷ�ֵ
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);// ��Χ�е���С�������
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	// �߳������������ϵ�����
    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if ((x >= imageW) || (y >= imageH)) return;
	// �����ƽ����(-1,-1,-1)��(1,1,1)����ά�ռ�
    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
	// ������������ϵ�е�����
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	// ��ʼ���߷���(u,v,-2λ��)
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
	// ���ӽǵ����߷���
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
	// ͨ����Χ���㷨���������Ƿ����������ཻ
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane// ������С��0����Ϊ0

    // march along ray from front to back, accumulating color
	// ������ߴ�ǰ���������ɫ
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;//��ʼλ��
    float3 step = eyeRay.d*tstep;

    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        //sample *= 64.0f;    // scale for 10-bit data

        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= brightness;
	// ����ɫֵд���������
    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

// ���������˲�ģʽ
extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}


// ��ʼ��CUDA
extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
	// ������ά����
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));// ����GPU�ڴ�
	// �������ݵ�GPU
    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));



	// �����������
    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates//ʹ�ñ�׼��������ϵ
    tex.filterMode = hipFilterModeLinear;      // linear interpolation//���Բ�ֵ�˲�
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;
	//�����鵽3D����
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
	//����ת����������������������������ݵ���ɫ��ÿһ��float4Ϊȡ�����RGBA
    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };
	// ��1D�������ʽ����ɫת������󶨵�transferTex
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));
	// ����3D����Ĳ���
    transferTex.filterMode = hipFilterModeLinear;// ��ֵ��ʽ�����Բ�ֵ
    transferTex.normalized = true;    // access with normalized texture coordinates����һ���������꣺true
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates//��ȡ����Ѱַ

	// ������󶨵�������
    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}
// �ͷ�CUDA����
extern "C"
void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
    checkCudaErrors(hipFreeArray(d_transferFuncArray));
	
}



// �����ӽǾ���
extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}
//////////////////////////////////////////////////////////////////////////////////////
//���������ڴ洢��������Ż�
//
//
//
//
//
//////////////////////////////////////////////////////////////////////////////////////

__global__ void
my_d_render(
uint *d_output, //�������
uint imageW, //ͼ��Ŀ��
uint imageH,//ͼ��ĸ߶�
         float density, //ͼ���ܶ�
		 float brightness,//ͼ������
         float transferOffset, //����
		 float transferScale,//�����Ϳ̶�
		 hipPitchedPtr devPitchedPtr,
		 hipExtent devextent
		 )
{
    const int maxSteps = 500;
    const float tstep = 0.01f;// ����
    const float opacityThreshold = 0.95f;// ͸���ȷ�ֵ
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);// ��Χ�е���С�������
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	char * devPtr = (char *)devPitchedPtr.ptr;
	size_t pitch = devPitchedPtr.pitch;
	size_t slicePitch = pitch*devextent.height;

	// �߳������������ϵ�����
    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;
	uint z = blockIdx.z*blockDim.z + threadIdx.z;

    if ((x >= imageW) || (y >= imageH)) return;
	// �����ƽ����(-1,-1,-1)��(1,1,1)����ά�ռ�
    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
	// ������������ϵ�е�����
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	// ��ʼ���߷���(u,v,-2λ��)
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
	// ���ӽǵ����߷���
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
	// ͨ����Χ���㷨���������Ƿ����������ཻ
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane// ������С��0����Ϊ0

    // march along ray from front to back, accumulating color
	// ������ߴ�ǰ���������ɫ
    float4 sum = make_float4(0.0f);
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;//��ʼλ��
    float3 step = eyeRay.d*tstep;

    for (int i=0; i<maxSteps; i++)
    {
        // read from 3D texture
        // remap position to [0, 1] coordinates
        //float sample = tex3D(tex, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);
        //sample *= 64.0f;    // scale for 10-bit data
		char * slice = devPtr+z*slicePitch;
		float * row = (float *)(slice + y*pitch);
		float element = row[x];


		float sample = 0.3;


        // lookup in transfer function texture
        float4 col = tex1D(transferTex, (sample-transferOffset)*transferScale);
        col.w *= density;

        // "under" operator for back-to-front blending
        //sum = lerp(sum, col, col.w);

        // pre-multiply alpha
        col.x *= col.w;
        col.y *= col.w;
        col.z *= col.w;
        // "over" operator for front-to-back blending
        sum = sum + col*(1.0f - sum.w);

        // exit early if opaque
        if (sum.w > opacityThreshold)
            break;

        t += tstep;

        if (t > tfar) break;

        pos += step;
    }

    sum *= brightness;
	// ����ɫֵд���������
    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(sum);
}

// ����ȫ�ִ洢������ ��ʼ��CUDA
extern "C"
void myinitCuda(void *h_volume, hipExtent volumeSize)
{
	checkCudaErrors(hipMalloc3D(&devPitchedPtr,volumeSize));

	//devPitchedPtr=make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
	hipMemcpy3DParms mycopyParams = {0};
    mycopyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    mycopyParams.dstPtr = devPitchedPtr;
    mycopyParams.extent   = volumeSize;
    mycopyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&mycopyParams));

	devextent = volumeSize;


	//hipFree((void *)&devPitchedPtr);


	// ������ά����
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));// ����GPU�ڴ�
	// �������ݵ�GPU
    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));


	/*
	// �����������
    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates//ʹ�ñ�׼��������ϵ
    tex.filterMode = hipFilterModeLinear;      // linear interpolation//���Բ�ֵ�˲�
    tex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    tex.addressMode[1] = hipAddressModeClamp;
	//�����鵽3D����
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
	//*/
	//����ת����������������������������ݵ���ɫ��ÿһ��float4Ϊȡ�����RGBA
    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };
	// ��1D�������ʽ����ɫת������󶨵�transferTex
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));
	// ����3D����Ĳ���
    transferTex.filterMode = hipFilterModeLinear;// ��ֵ��ʽ�����Բ�ֵ
    transferTex.normalized = true;    // access with normalized texture coordinates����һ���������꣺true
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates//��ȡ����Ѱַ

	// ������󶨵�������
    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));
}

// ����CUDA�ں˽��������
extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
                   float density, float brightness, float transferOffset, float transferScale)
{

    //my_d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
    //                                  brightness, transferOffset, transferScale,devPitchedPtr,devextent);
	d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                      brightness, transferOffset, transferScale);
}



#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
